
#include <hip/hip_runtime.h>
void sync_gpu_buffer(void* dst, const void* src, size_t count)
{
	hipMemcpy ( dst, src, count, hipMemcpyHostToDevice);
}

void sync_cpu_buffer(void* dst, const void* src, size_t count)
{
	hipMemcpy ( dst, src, count, hipMemcpyDeviceToHost);
}