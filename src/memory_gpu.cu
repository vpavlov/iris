#include "hip/hip_runtime.h"
// -*- c++ -*-
//==============================================================================
// IRIS - Long-range Interaction Solver Library
//
// Copyright (c) 2017-2020, the National Center for Supercomputing Applications
//
// Primary authors:
//     Valentin Pavlov <vpavlov@rila.bg>
//     Peicho Petkov <peicho@phys.uni-sofia.bg>
//     Stoyan Markov <markov@acad.bg>
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
// THE SOFTWARE.
//==============================================================================
#include <stdlib.h>
#include <string.h>
#include <malloc.h>
#include <new>
#include <cstdint>
#include "memory.h"
#include "utils.h"
#include "memory_gpu.cuh"
#include "cuda_parameters.h"

using namespace ORG_NCSA_IRIS;

std::map<void *, std::map<std::string, void*> > memory_gpu::gpu_allocated_pointers;
std::map<void *, std::array<int,3> > memory_gpu::gpu_allocated_pointers_shape;

void *memory_gpu::wmalloc(int nbytes, void * parent,  const std::string label)
{
    void *retval = NULL;
    
    if (!label.empty()) {
        retval = get_registered_gpu_pointer(parent, label);
    }

    if (retval==NULL || label.empty()) {
           HANDLE_LAST_CUDA_ERROR;
    hipError_t res = hipMalloc((void**)&retval, nbytes);
    HANDLE_LAST_CUDA_ERROR;
    if(res != hipSuccess) {
	throw std::bad_alloc();
    }
    if(!label.empty()) {
        register_gpu_pointer(parent,label,retval);
    }
    }

    return retval;
};

void *memory_gpu::wrealloc(void *ptr, int nbytes, int old_size)
{
    if(nbytes == 0) {
	wfree(ptr);
	return NULL;
    }

    void *tmp = wmalloc(nbytes);
    hipMemcpy(tmp, ptr, MIN(nbytes,old_size),hipMemcpyDeviceToDevice);
    HANDLE_LAST_CUDA_ERROR;
	wfree(ptr);
	return tmp;
 };

void memory_gpu::wfree(void *ptr, bool keep_it)
{
    if ((!keep_it) && (ptr!=NULL)) {
    HANDLE_LAST_CUDA_ERROR;
    hipFree(ptr);
    HANDLE_LAST_CUDA_ERROR;
    auto pl = get_parent_and_label(ptr);
    unregister_gpu_pointer(pl.first,pl.second);
    unregister_gpu_pointer_shape(ptr);
    }
};


__global__
void memory_set_kernel(iris_real* ptr, int n, iris_real val)
{
    int ndx = IRIS_CUDA_INDEX(x);
    int chunk_size = IRIS_CUDA_CHUNK(x,n);
    int from = ndx*chunk_size;
    int to = MIN((ndx+1)*chunk_size,n);
    
    for(ndx=from; ndx<to; ++ndx) {
    ptr[ndx]=val;
    }
};

__global__
void print_kernel(iris_real* ptr, int n, const char* name)
{
    int ndx = IRIS_CUDA_INDEX(x);
    int chunk_size = IRIS_CUDA_CHUNK(x,n);
    int from = ndx*chunk_size;
    int to = MIN((ndx+1)*chunk_size,n);
    
    for(ndx=from; ndx<to; ++ndx) {
        if(ndx==0)
    printf("%s[%d] %f",name,ndx,ptr[ndx]);
    }
};

void print_vector_gpu(iris_real* ptr, int n, const char* name)
{   
    int blocks = get_NBlocks_X(n,IRIS_CUDA_NTHREADS_YX);
    int threads = MIN((n+blocks+1)/blocks,IRIS_CUDA_NTHREADS_YX);
    print_kernel<<<blocks,threads>>>(ptr,n,name);
    hipDeviceSynchronize();
    HANDLE_LAST_CUDA_ERROR;
};


__global__
void memory_set_kernel(iris_real*** ptr3d, int n, iris_real val)
{
    iris_real *ptr = &(ptr3d[0][0][0]);
    int ndx = IRIS_CUDA_INDEX(x);
    int chunk_size = IRIS_CUDA_CHUNK(x,n);
    int from = ndx*chunk_size;
    int to = MIN((ndx+1)*chunk_size,n);
    
    for(ndx=from; ndx<to; ++ndx) {
    ptr[ndx]=val;
    }
};

__global__
void memory_set_kernel(iris_real** ptr2d, int n, iris_real val)
{
    iris_real *ptr = &(ptr2d[0][0]);
    int ndx = IRIS_CUDA_INDEX(x);
    int chunk_size = IRIS_CUDA_CHUNK(x,n);
    int from = ndx*chunk_size;
    int to = MIN((ndx+1)*chunk_size,n);
    
    for(ndx=from; ndx<to; ++ndx) {
    ptr[ndx]=val;
    }
};

//**********************************************************************
// 1D Arrays
//**********************************************************************

iris_real *memory_gpu::create_1d(iris_real *&array, int n1, bool clear,
                                    void * parent,  const std::string label)
{

    if ((!has_shape((void*)array,{n1,0,0}))&&(!label.empty())) {
        wfree(array);
    }

    array =  (iris_real *)wmalloc(sizeof(iris_real) * n1, parent, label);
    
    if(!label.empty()) {
    register_gpu_pointer_shape((void*)array,{n1,0,0});
    }

    if(clear) {
      int blocks = get_NBlocks_X(n1,IRIS_CUDA_NTHREADS_YX);
      int threads = IRIS_CUDA_NTHREADS_YX;
      memory_set_kernel<<<blocks,threads>>>(array,n1,(iris_real)0);
      hipDeviceSynchronize();
      HANDLE_LAST_CUDA_ERROR;
    }
    return array;
};


void memory_gpu::destroy_1d(iris_real *&array, bool keep_it)
{
    if(array == NULL) {
	return;
	}

    if(keep_it) {
    return;
    }
	
    wfree(array);
	array = NULL;
};

////////////////////////////////////////////

__global__
void assign_2d_indexing_kernel(iris_real** array,iris_real* tmp, int n1, int n2)
{

    int xndx = IRIS_CUDA_INDEX(x);
    int xchunk_size = IRIS_CUDA_CHUNK(x,n1);
    int xfrom = xndx*xchunk_size;
    int xto = MIN((xndx+1)*xchunk_size,n1);

    for (xndx=xfrom; xndx<xto; ++xndx) {
        int m = xndx*n2;
        array[xndx]=&tmp[m];
    }
};


//**********************************************************************
// 2D Arrays
//**********************************************************************

iris_real **memory_gpu::create_2d(iris_real **&array, int n1, int n2, bool clear, 
								void * parent,  const std::string label)
{
    if((!has_shape((void*)array,{n1,n2,0}))&&(!label.empty())) {
        destroy_2d(array);
    }

    int nitems = n1 * n2;

    void* ptr = get_registered_gpu_pointer(parent,label);
    
    if (ptr==NULL) {
        array =  (iris_real **)wmalloc(sizeof(iris_real *) * n1, parent, label);
        iris_real* data = (iris_real *)wmalloc(sizeof(iris_real) * nitems);
        assign_2d_indexing_kernel<<<get_NBlocks_X(n1,IRIS_CUDA_NTHREADS_YX),IRIS_CUDA_NTHREADS_YX>>>(array,data,n1,n2);
        register_gpu_pointer_shape(array,{n1,n2,0});
    } else {
        array = (iris_real **)ptr;
    }

    if(clear) {
        memory_set_kernel<<<get_NBlocks_X(nitems,IRIS_CUDA_NTHREADS_YX),IRIS_CUDA_NTHREADS_YX>>>(array,nitems,(iris_real)0);
    }

    
    hipDeviceSynchronize();
    HANDLE_LAST_CUDA_ERROR;

    if(!label.empty()) {
    register_gpu_pointer_shape((void*)array,{n1,n2,0});
    }

    return array;
};

__global__
void get_2d_1d_pointer_kernel(iris_real **prt, iris_real *&ptr1d)
{
    ptr1d = prt[0];
}


void memory_gpu::destroy_2d(iris_real **&array, bool keep_it)
{
    if(array == NULL) {
    return;
    }

    if(!keep_it) {
    return;
    }

    iris_real *data;
    get_2d_1d_pointer_kernel<<<1,1>>>(array,data);
    hipDeviceSynchronize();
    HANDLE_LAST_CUDA_ERROR;
    wfree(data);
    wfree(array);     // free the array
    array = NULL;
};

/////////////////////////////////////////////////////////

__global__
void assign_3d_indexing_kernel(iris_real*** array, iris_real** tmp, iris_real* data, int n1, int n2, int n3)
{
    int xndx = IRIS_CUDA_INDEX(x);
    int xchunk_size = IRIS_CUDA_CHUNK(x,n1);
    int yndx = IRIS_CUDA_INDEX(y);
    int ychunk_size = IRIS_CUDA_CHUNK(y,n2);

    int xfrom = xndx*xchunk_size;
    int xto = MIN((xndx+1)*xchunk_size,n1);

    int yfrom = yndx*ychunk_size;
    int yto = MIN((yndx+1)*ychunk_size,n2);
    int m,n;
    for (int i=xfrom; i<xto; ++i) {
        m = i*n2;
        if (yfrom==0){
        array[i]=&tmp[m];
        //printf("xfrom %d xto %d array[%d] &tmp[%d]\n",xfrom,xto,i,m);
        }
        for (int j=yfrom; j<yto; ++j) {
            n = (m+j)*n3;
            tmp[m+j] = &data[n];
            //printf("tmp[%d] &data[%d] (i+1) %d j %d n3 %d\n",m+j,n,i+1,j,n3);
        }
    }
}


//**********************************************************************
// 3D Arrays
//**********************************************************************

iris_real ***memory_gpu::create_3d(iris_real ***&array, int n1, int n2, int n3,
                bool clear, iris_real init_val, void * parent,  const std::string label)
{
    int nitems = n1 * n2 * n3;

    if((!has_shape(array,{n1,n2,n3}))&&(!label.empty())) {
        destroy_3d(array);
    }

    void* ptr = get_registered_gpu_pointer(parent,label);
    if (ptr==NULL) {
        array   = (iris_real ***) wmalloc(sizeof(iris_real **) * n1,parent,label);
        iris_real **tmp = (iris_real **)  wmalloc(sizeof(iris_real *)  * n1 * n2);
        iris_real *data = (iris_real *)   wmalloc(sizeof(iris_real)    * nitems);
        int nblocks1 = get_NBlocks_X(n1,IRIS_CUDA_NTHREADS_YX);
        int nblocks2 = get_NBlocks_YZ(n2,IRIS_CUDA_NTHREADS_YX);
        int nthreads1 = IRIS_CUDA_NTHREADS_YX;
        int nthreads2 = IRIS_CUDA_NTHREADS_YX;
        assign_3d_indexing_kernel<<<dim3(nblocks1,nblocks2),dim3(nthreads1,nthreads2)>>>(array, tmp, data, n1, n2, n3);
        hipDeviceSynchronize();
        HANDLE_LAST_CUDA_ERROR;
        if(!label.empty()) {
        register_gpu_pointer(parent,label,array);
        }
    } else {
        array = (iris_real***) ptr;
    }

    if(clear) {
      int blocks = get_NBlocks_X(nitems,IRIS_CUDA_NTHREADS_YX);
      int threads = IRIS_CUDA_NTHREADS_YX;
      memory_set_kernel<<<blocks,threads>>>(array,nitems, init_val);
    }
    hipDeviceSynchronize();
    HANDLE_LAST_CUDA_ERROR;

    if(!label.empty()) {
    register_gpu_pointer_shape((void*)array,{n1,n2,0});
    }

    return array;
};


iris_real ***memory_gpu::create_3d(iris_real ***&array, int n1, int n2, int n3,
                bool clear, iris_real init_val, iris_real *&data, void * parent,  const std::string label)
{
    int nitems = n1 * n2 * n3;

    if((!has_shape(array,{n1,n2,n3}))&&(!label.empty())) {
        destroy_3d(array);
    }

    void* ptr = get_registered_gpu_pointer(parent,label);
    if (ptr==NULL) {
        array   = (iris_real ***) wmalloc(sizeof(iris_real **) * n1,parent,label);
        iris_real **tmp = (iris_real **)  wmalloc(sizeof(iris_real *)  * n1 * n2);
        data = (iris_real *)   wmalloc(sizeof(iris_real)    * nitems);
        int nblocks1 = get_NBlocks_X(n1,IRIS_CUDA_NTHREADS_YX);
        int nblocks2 = get_NBlocks_YZ(n2,IRIS_CUDA_NTHREADS_YX);
        int nthreads1 = IRIS_CUDA_NTHREADS_YX;
        int nthreads2 = IRIS_CUDA_NTHREADS_YX;
        assign_3d_indexing_kernel<<<dim3(nblocks1,nblocks2),dim3(nthreads1,nthreads2)>>>(array, tmp, data, n1, n2, n3);
        hipDeviceSynchronize();
        HANDLE_LAST_CUDA_ERROR;
        if(!label.empty()) {
        register_gpu_pointer(parent,label,array);
        }
    } else {
        array = (iris_real***) ptr;
    }

    if(clear) {
      int blocks = get_NBlocks_X(nitems,IRIS_CUDA_NTHREADS_YX);
      int threads = IRIS_CUDA_NTHREADS_YX;
      memory_set_kernel<<<blocks,threads>>>(array,nitems, init_val);
    }
    hipDeviceSynchronize();
    HANDLE_LAST_CUDA_ERROR;

    if(!label.empty()) {
    register_gpu_pointer_shape((void*)array,{n1,n2,0});
    }

    return array;
};

__global__
void get_3d_2d_1d_pointer_kernel(iris_real ***ptr3d,iris_real **&ptr2d, iris_real *&ptr1d)
{
    ptr2d = ptr3d[0];
    ptr1d = ptr3d[0][0];
}

void memory_gpu::destroy_3d(iris_real ***&array, bool keep_it)
{
  #warning "not sure if it really free the allocated mamory"
    if(array == NULL) {
    return;
    }

    if(keep_it) {
    return;
    }
    //size_t free, total;

    //printf("hipMemGetInfo LAST CUDA EROOR: %s\n",hipGetErrorString ( hipGetLastError()  ));
    //wfree((void*)&array[0][0][0]);
    //printf("(void*)&array[0][0][0] LAST CUDA EROOR: %s\n",hipGetErrorString ( hipGetLastError()  ));
    
    iris_real **tmpmap;
    iris_real *datap;
    get_3d_2d_1d_pointer_kernel<<<1,1>>>(array,tmpmap,datap);
    hipDeviceSynchronize();
    HANDLE_LAST_CUDA_ERROR;

    wfree(datap);
    wfree(tmpmap);
    wfree(array);
        
    //printf("array LAST CUDA EROOR: %s\n",hipGetErrorString ( hipGetLastError()  ));
    //hipMemGetInfo(&free,&total);
    //printf("free %d total %d\n");

    array = NULL;
};

__global__
void get_data_pointer(iris_real ***array, iris_real **data_p)
{
    *data_p=&(array[0][0][0]);
}

int memory_gpu::sync_gpu_buffer(iris_real*** dst_gpu, void* src, size_t count)
{
    iris_real **gpu_data_p;
    iris_real *tmp;
    hipMalloc(&gpu_data_p,sizeof(iris_real *));
    get_data_pointer<<<1,1>>>(dst_gpu,gpu_data_p);
    hipDeviceSynchronize();
    HANDLE_LAST_CUDA_ERROR;
    hipMemcpy ( &tmp, gpu_data_p, sizeof(iris_real *), hipMemcpyDeviceToHost);
    hipFree(gpu_data_p);
    return sync_gpu_buffer(tmp,src,count);
}

int memory_gpu::sync_cpu_buffer(void* dst, iris_real*** src_gpu, size_t count)
{
	iris_real **gpu_data_p;
    iris_real *tmp;
    hipMalloc(&gpu_data_p,sizeof(iris_real *));
    get_data_pointer<<<1,1>>>(src_gpu,gpu_data_p);
    hipDeviceSynchronize();
    HANDLE_LAST_CUDA_ERROR;
    hipMemcpy ( &tmp, gpu_data_p, sizeof(iris_real *), hipMemcpyDeviceToHost);
    hipFree(gpu_data_p);
    return sync_cpu_buffer(dst,tmp,count);
}

int memory_gpu::sync_gpu_buffer(void* dst_gpu, const void* src, size_t count)
{
	return hipMemcpy ( dst_gpu, src, count, hipMemcpyHostToDevice);
}

int memory_gpu::sync_cpu_buffer(void* dst, const void* src_gpu, size_t count)
{
	return hipMemcpy ( dst, src_gpu, count, hipMemcpyDeviceToHost);
}

void * memory_gpu::get_registered_gpu_pointer(void *parent, std::string label)
{
    auto it = gpu_allocated_pointers.find(parent);
    if (it!=gpu_allocated_pointers.end()) {
        auto it1 = it->second.find(label);
        if (it1!=it->second.end()) {
            return it1->second;
        }
    }
    return NULL;
}

void memory_gpu::register_gpu_pointer(void *parent, std::string label, void* ptr)
{
    gpu_allocated_pointers[parent][label]=ptr;
}

void memory_gpu::unregister_gpu_pointer(void *parent, std::string label)
{
    auto it = gpu_allocated_pointers.find(parent);
    if (it!=gpu_allocated_pointers.end()) {
        auto it1 = it->second.find(label);
        if (it1!=it->second.end()) {
            it->second.erase(label);
        }
        if (it->second.empty())
        {
            gpu_allocated_pointers.erase(it);
        }
    }
}

bool memory_gpu::has_shape(void *ptr, std::array<int,3> in_shape)
{
    auto it = gpu_allocated_pointers_shape.find(ptr);
    if (it!=gpu_allocated_pointers_shape.end()) {
        if (it->second==in_shape) {
            return true;
        }
    }
    return false;
}

void memory_gpu::register_gpu_pointer_shape(void *ptr, std::array<int,3> in_shape)
{
    gpu_allocated_pointers_shape[ptr]=in_shape;
}

void memory_gpu::unregister_gpu_pointer_shape(void *ptr)
{
    auto it = gpu_allocated_pointers_shape.find(ptr);
    if (it!=gpu_allocated_pointers_shape.end()) {
        gpu_allocated_pointers_shape.erase(it);
    }
}

std::pair<void *,std::string> memory_gpu::get_parent_and_label(void* prt)
{
    for (auto it=gpu_allocated_pointers.begin(); it!=gpu_allocated_pointers.end();it++) {
            for (auto entry_it=it->second.begin(); entry_it!=it->second.end();entry_it++)
            {
                if (entry_it->second==prt)
                {
                    return std::pair<void*, std::string>(it->first,entry_it->first);
                }
            }
    }
    return std::pair<void*, std::string>(NULL,"");
}


const int BLOCK_SIZE = IRIS_CUDA_NTHREADS_Z*IRIS_CUDA_NTHREADS_Z*IRIS_CUDA_NTHREADS_Z;

__global__
void dot_kernel(iris_real ***in_rho_phi,
                        iris_real *dot,
                        int nx, int ny, int nz)
{
    __shared__ iris_real Ek_acc[BLOCK_SIZE];
    
    int xndx = IRIS_CUDA_INDEX(x);
    int xchunk_size = IRIS_CUDA_CHUNK(x,nx);
    int yndx = IRIS_CUDA_INDEX(y);
    int ychunk_size = IRIS_CUDA_CHUNK(y,ny);
    int zndx = IRIS_CUDA_INDEX(z);
    int zchunk_size = IRIS_CUDA_CHUNK(z,nz);

	int i_from = xndx*xchunk_size, i_to = MIN((xndx+1)*xchunk_size,nx);
	int j_from = yndx*ychunk_size, j_to = MIN((yndx+1)*ychunk_size,ny);
	int k_from = zndx*zchunk_size, k_to = MIN((zndx+1)*zchunk_size,nz);

    int iacc = xndx*IRIS_CUDA_NTHREADS_Z*IRIS_CUDA_NTHREADS_Z + yndx*IRIS_CUDA_NTHREADS_Z + zndx;



    // printf("bdimx %d bdimy %d bdimz %d tidx %d tidy %d tidz %d\n",blockDim.x,blockDim.y,blockDim.z, threadIdx.x, threadIdx.y,threadIdx.z);

    //printf("iacc = %d i_from %d i_to %d j_from %d j_to %d k_from %d k_to %d xchunk_size %d ychunk_size %d zchunk_size %d nx %d ny %d nz %d\n",iacc,i_from,i_to,j_from,j_to,k_from,k_to, xchunk_size, ychunk_size, zchunk_size,nx,ny,nz);
    //printf("out of the for bdimx %d bdimy %d bdimz %d tidx %d tidy %d tidz %d ndx %d %d %d i_from %d i_to %d j_from %d j_to %d k_from %d k_to %d\n",blockDim.x,blockDim.y,blockDim.z, threadIdx.x, threadIdx.y,threadIdx.z,xndx,yndx,zndx,i_from,i_to,j_from,j_to,k_from,k_to);
   

    Ek_acc[iacc] = 0.0;

  // printf("bdimx %d bdimy %d bdimz %d tidx %d tidy %d tidz %d\n",blockDim.x,blockDim.y,blockDim.z, threadIdx.x, threadIdx.y,threadIdx.z);

        for(int i=i_from;i<i_to;i++) {
            int ni = i*ny*nz;
            for(int j=j_from;j<j_to;j++) {
                int nj = ni + j*nz;
                for(int k=k_from;k<k_to;k++) {
                int n = nj + k;
                Ek_acc[iacc] += in_rho_phi[i][j][k]*in_rho_phi[i][j][k];
                }
            }
        }

    __syncthreads();

    for(int i = BLOCK_SIZE; i > 1; i/=2 ) {
        //printf("echo  BLOCK_SIZE %d ibl %d iacc %d + %d BLOCK_SIZE/i %d (iacc)%(BLOCK_SIZE/i) %d \n",BLOCK_SIZE,i,iacc,iacc+BLOCK_SIZE/i,BLOCK_SIZE/i,(iacc)%(BLOCK_SIZE/i));
        int stride = BLOCK_SIZE/i;
        if (iacc < (BLOCK_SIZE - stride)  && (iacc)%(2*stride)==0) {
          //  printf("i %d Ek_acc[%d] %f Ek_acc[%d] %f\n", i, iacc,Ek_acc[iacc],iacc+stride,Ek_acc[iacc+stride]);
            Ek_acc[iacc] += Ek_acc[iacc+stride];
            //printf("echo ibl %d iacc %d + %d BLOCK_SIZE/i %d (iacc)%(BLOCK_SIZE/i) %d \n",i,iacc,iacc+BLOCK_SIZE/i,BLOCK_SIZE/i,(iacc)%(BLOCK_SIZE/i));
            
       }
        __syncthreads();
    }

    if (iacc==0) {
        atomicAdd(dot,Ek_acc[iacc]);
    }
}

__global__
void sum_kernel(iris_real ***in_rho_phi,
                        iris_real *dot,
                        int nx, int ny, int nz)
{
    __shared__ iris_real Ek_acc[BLOCK_SIZE];
    
    int xndx = IRIS_CUDA_INDEX(x);
    int xchunk_size = IRIS_CUDA_CHUNK(x,nx);
    int yndx = IRIS_CUDA_INDEX(y);
    int ychunk_size = IRIS_CUDA_CHUNK(y,ny);
    int zndx = IRIS_CUDA_INDEX(z);
    int zchunk_size = IRIS_CUDA_CHUNK(z,nz);

	int i_from = xndx*xchunk_size, i_to = MIN((xndx+1)*xchunk_size,nx);
	int j_from = yndx*ychunk_size, j_to = MIN((yndx+1)*ychunk_size,ny);
	int k_from = zndx*zchunk_size, k_to = MIN((zndx+1)*zchunk_size,nz);

    int iacc = xndx*IRIS_CUDA_NTHREADS_Z*IRIS_CUDA_NTHREADS_Z + yndx*IRIS_CUDA_NTHREADS_Z + zndx;



    // printf("bdimx %d bdimy %d bdimz %d tidx %d tidy %d tidz %d\n",blockDim.x,blockDim.y,blockDim.z, threadIdx.x, threadIdx.y,threadIdx.z);

    //printf("iacc = %d i_from %d i_to %d j_from %d j_to %d k_from %d k_to %d xchunk_size %d ychunk_size %d zchunk_size %d nx %d ny %d nz %d\n",iacc,i_from,i_to,j_from,j_to,k_from,k_to, xchunk_size, ychunk_size, zchunk_size,nx,ny,nz);
    //printf("out of the for bdimx %d bdimy %d bdimz %d tidx %d tidy %d tidz %d ndx %d %d %d i_from %d i_to %d j_from %d j_to %d k_from %d k_to %d\n",blockDim.x,blockDim.y,blockDim.z, threadIdx.x, threadIdx.y,threadIdx.z,xndx,yndx,zndx,i_from,i_to,j_from,j_to,k_from,k_to);
  
    Ek_acc[iacc] = 0.0;

  // printf("bdimx %d bdimy %d bdimz %d tidx %d tidy %d tidz %d\n",blockDim.x,blockDim.y,blockDim.z, threadIdx.x, threadIdx.y,threadIdx.z);

        for(int i=i_from;i<i_to;i++) {
            int ni = i*ny*nz;
            for(int j=j_from;j<j_to;j++) {
                int nj = ni + j*nz;
                for(int k=k_from;k<k_to;k++) {
                int n = nj + k;
                Ek_acc[iacc] += in_rho_phi[i][j][k];
                }
            }
        }

    __syncthreads();

    for(int i = BLOCK_SIZE; i > 1; i/=2 ) {
        //printf("echo  BLOCK_SIZE %d ibl %d iacc %d + %d BLOCK_SIZE/i %d (iacc)%(BLOCK_SIZE/i) %d \n",BLOCK_SIZE,i,iacc,iacc+BLOCK_SIZE/i,BLOCK_SIZE/i,(iacc)%(BLOCK_SIZE/i));
        int stride = BLOCK_SIZE/i;
        if (iacc < (BLOCK_SIZE - stride)  && (iacc)%(2*stride)==0) {
          //  printf("i %d Ek_acc[%d] %f Ek_acc[%d] %f\n", i, iacc,Ek_acc[iacc],iacc+stride,Ek_acc[iacc+stride]);
            Ek_acc[iacc] += Ek_acc[iacc+stride];
            //printf("echo ibl %d iacc %d + %d BLOCK_SIZE/i %d (iacc)%(BLOCK_SIZE/i) %d \n",i,iacc,iacc+BLOCK_SIZE/i,BLOCK_SIZE/i,(iacc)%(BLOCK_SIZE/i));
            
       }
        __syncthreads();
    }

    if (iacc==0) {
        atomicAdd(dot,Ek_acc[iacc]);
    }
}


iris_real calc_sum(iris_real ***v, int nx, int ny, int nz)
{
    int nthreads1 = IRIS_CUDA_NTHREADS_Z;
	int nthreads2 = IRIS_CUDA_NTHREADS_Z;
	int nthreads3 = IRIS_CUDA_NTHREADS_Z;
    int nblocks1 = get_NBlocks_X(nx,IRIS_CUDA_NTHREADS_Z);
	int nblocks2 = get_NBlocks_YZ(ny,IRIS_CUDA_NTHREADS_Z);
	int nblocks3 = get_NBlocks_YZ(nz,IRIS_CUDA_NTHREADS_Z);

	auto blocks = dim3(nblocks1,nblocks2,nblocks3);
    auto threads = dim3(nthreads1,nthreads2,nthreads3);
    iris_real *res;
    iris_real result;
    hipMalloc((void**)&res,sizeof(iris_real));
     HANDLE_LAST_CUDA_ERROR;
    sum_kernel<<<blocks,threads>>>(v,res,nx,ny,nz);
    hipDeviceSynchronize();
    HANDLE_LAST_CUDA_ERROR;
    hipMemcpy ( &result, res,sizeof(iris_real), hipMemcpyDeviceToHost);
     HANDLE_LAST_CUDA_ERROR;
    return result;
}