
#include <hip/hip_runtime.h>
// -*- c++ -*-
//==============================================================================
// IRIS - Long-range Interaction Solver Library
//
// Copyright (c) 2017-2021, the National Center for Supercomputing Applications
//
// Primary authors:
//     Valentin Pavlov <vpavlov@rila.bg>
//     Peicho Petkov <peicho@phys.uni-sofia.bg>
//     Stoyan Markov <markov@acad.bg>
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
// THE SOFTWARE.
//==============================================================================
#ifdef IRIS_CUDA
#include <assert.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include "cuda.h"
#include "comm_rec.h"
#include "fmm.h"
#include "real.h"
#include "ses.h"

using namespace ORG_NCSA_IRIS;


////////////////////
// Load Particles //
////////////////////


__global__ void k_load_charges(iris_real *charges, int ncharges, int hwm,
			       iris_real xlo, iris_real ylo, iris_real zlo,
			       iris_real lsx, iris_real lsy, iris_real lsz,
			       int max_level, int offset, particle_t *m_particles, int rank,
			       int *atom_types)
{
    IRIS_CUDA_SETUP_WS(ncharges);
    for(int i=from;i<to;i++) {
	iris_real tx = (charges[i * 5 + 0] - xlo) / lsx;
	iris_real ty = (charges[i * 5 + 1] - ylo) / lsy;
	iris_real tz = (charges[i * 5 + 2] - zlo) / lsz;

	int lc[3];
	lc[0] = (int) tx;
	lc[1] = (int) ty;
	lc[2] = (int) tz;
	
	int id = 0;
	for(int l=0;l<max_level; l++) {
	    for(int d=0;d<3;d++) {
		id += (lc[d] & 1) << (3*l + d);
		lc[d] >>= 1;
	    }
	}
	
	int cellID = offset + id;
	int chargeID = (int)charges[i*5 + 4];
	
	m_particles[i+hwm].rank = rank;
	m_particles[i+hwm].index = chargeID; // (chargeID > 0)?chargeID:-chargeID;
	m_particles[i+hwm].cellID = cellID;
	m_particles[i+hwm].xyzq[0] = charges[i*5+0];
	m_particles[i+hwm].xyzq[1] = charges[i*5+1];
	m_particles[i+hwm].xyzq[2] = charges[i*5+2];
	m_particles[i+hwm].xyzq[3] = charges[i*5+3];
	m_particles[i+hwm].tgt[0] = 0.0;
	m_particles[i+hwm].tgt[1] = 0.0;
	m_particles[i+hwm].tgt[2] = 0.0;
	m_particles[i+hwm].tgt[3] = 0.0;
	atom_types[i+hwm] = (chargeID > 0)?0:1;
    }
}

__global__ void k_extract_cellID(particle_t *m_particles, int n, int *cellID_keys)
{
    IRIS_CUDA_SETUP_WS(n);
    for(int i=from;i<to;i++) {
	cellID_keys[i] = m_particles[i].cellID;
    }
}

void fmm::load_particles_gpu()
{
    // Find the total amount of local charges, including halo atoms
    // This is just a simple sum of ncharges from all incoming rank
    int total_local_charges = 0;
    for(int rank = 0; rank < m_iris->m_client_size; rank++ ) {
	total_local_charges += m_iris->m_ncharges[rank];
    }

    // Allocate GPU memory for m_particles (and m_xparticles, it will be at the end of m_particles)
    // Also, allocate same sized array for the atom types (=1 if halo atom, =0 if own atom)
    m_particles = (particle_t *)memory::wmalloc_gpu_cap((void *)m_particles, total_local_charges, sizeof(particle_t), &m_npart_cap);
    m_atom_types = (int *)memory::wmalloc_gpu_cap((void *)m_atom_types, total_local_charges, sizeof(int), &m_at_cap);
    m_cellID_keys = (int *)memory::wmalloc_gpu_cap((void *)m_cellID_keys, total_local_charges, sizeof(int), &m_cellID_keys_cap);

    // Allocate GPU memory for the charges coming from all client ranks
    // This is done all at once so as no to interfere with mem transfer/kernel overlapping in the next loop
    for(int rank = 0; rank < m_iris->m_client_size; rank++ ) {
	int ncharges = m_iris->m_ncharges[rank];
	iris_real *charges = m_iris->m_charges[rank];
	m_charges_gpu[rank] = (iris_real *)memory::wmalloc_gpu_cap((void *)m_charges_gpu[rank], ncharges, 5*sizeof(iris_real), &m_charges_gpu_cap[rank]);
    }

    // Start the particle loading itself. The <H2D, kernel> pairs runs on separate streams and thus overlaps memory transfer for more than 1 source rank
    int offset = cell_meta_t::offset_for_level(max_level());
    int nd = 1 << max_level();
    int hwm = 0;    
    for(int rank = 0; rank < m_iris->m_client_size; rank++ ) {
	int ncharges = m_iris->m_ncharges[rank];
	iris_real *charges = m_iris->m_charges[rank];
	cudaMemcpyAsync(m_charges_gpu[rank], charges, ncharges * 5 * sizeof(iris_real), cudaMemcpyDefault, m_streams[rank % IRIS_CUDA_FMM_NUM_STREAMS]);
	
	int nthreads = IRIS_CUDA_NTHREADS;
	int nblocks = IRIS_CUDA_NBLOCKS(ncharges, nthreads);
	k_load_charges<<<nblocks, nthreads, 0, m_streams[rank % IRIS_CUDA_FMM_NUM_STREAMS]>>>(m_charges_gpu[rank], ncharges, hwm,
											      m_domain->m_global_box.xlo, m_domain->m_global_box.ylo, m_domain->m_global_box.zlo,
											      m_leaf_size[0], m_leaf_size[1], m_leaf_size[2],
											      max_level(), offset, m_particles, rank,
											      m_atom_types);
	cudaError_t err = cudaGetLastError();
	IRIS_CUDA_HANDLE_ERROR(err);
	hwm += ncharges;
    }
    
        
    cudaDeviceSynchronize();  // all k_load_charges kernels must have finished to have valid m_particles

    // At this point we have the m_particles filled up, with mixed halo/own atoms, etc.
    // We need to sort them according to the m_atom_type keys array and split into m_particles and m_xparticles
    thrust::device_ptr<int>         keys(m_atom_types);
    thrust::device_ptr<particle_t>  values(m_particles);
    thrust::sort_by_key(keys, keys+total_local_charges, values);

    // Now the first part of m_particles contains local atoms; second contains halo atoms
    // Number of local particles can be taken from iris: num_local_atoms
    // Number of halo particles is total_local_charges - num_local_atoms
    //m_nparticles = m_iris->num_local_atoms();
    m_nparticles = thrust::count(keys, keys+total_local_charges, 0);
    m_nxparticles = total_local_charges - m_nparticles;
    m_xparticles = m_particles + m_nparticles;

    // Get the cellIDs in the reordered array
    int nthreads = IRIS_CUDA_NTHREADS;
    int nblocks = IRIS_CUDA_NBLOCKS(total_local_charges, nthreads);
    k_extract_cellID<<<nblocks, nthreads>>>(m_particles, total_local_charges, m_cellID_keys);

    // We now need to sort the m_particles and m_xparticles arrays by cellID
    thrust::device_ptr<int>         keys2(m_cellID_keys);
    thrust::device_ptr<particle_t>  part(m_particles);
    thrust::device_ptr<particle_t>  xpart(m_xparticles);
    thrust::sort_by_key(keys2, keys2+m_nparticles, part);
    thrust::sort_by_key(keys2+m_nparticles, keys2+total_local_charges, xpart);
    
    m_logger->info("FMM/GPU: This rank owns %d + %d halo particles", m_nparticles, m_nxparticles);
}


//////////////////////////
// Distribute particles //
//////////////////////////


__device__ void d_compute_ses(particle_t *in_particles, int num_points, int first_child, cell_t *out_target)
{
    point_t points[2*IRIS_MAX_NCRIT];
    if(num_points > 2*IRIS_MAX_NCRIT) {
	asm("trap;");
    }
    for(int i=0;i<num_points;i++) {
    	points[i].r[0] = in_particles[first_child+i].xyzq[0];
    	points[i].r[1] = in_particles[first_child+i].xyzq[1];
    	points[i].r[2] = in_particles[first_child+i].xyzq[2];
    }
    ses_of_points(points, num_points, &(out_target->ses));
}

__global__ void k_distribute_particles(particle_t *in_particles, int in_count, int in_flags, cell_t *out_target, int offset, int nleafs)
{
    if(in_count == 0) {
	return;
    }
    
    int tid = IRIS_CUDA_TID;
    int cellID = offset + tid;
    float fract = (1.0*in_count)/nleafs;
    int from = (int)(fract * tid);
    int to = MIN((int)(fract * (tid + 1)), in_count-1);

    while(from > 0 && in_particles[from].cellID >= cellID)       { from--; }
    while(from < in_count && in_particles[from].cellID < cellID) { from++; }
    while(to < in_count-1 && in_particles[to].cellID <= cellID)  { to++; }
    while(to >= 0 && in_particles[to].cellID > cellID)           { to--; }

    int num_children = (to - from + 1);
    if(num_children <= 0) {
	return;
    }

    out_target[cellID].first_child = from;
    out_target[cellID].num_children = num_children;
    out_target[cellID].flags = in_flags;
    d_compute_ses(in_particles, num_children, from, out_target+cellID);
}

void fmm::distribute_particles_gpu(struct particle_t *in_particles, int in_count, int in_flags, struct cell_t *out_target)
{
    int nleafs = (1 << 3 * max_level());
    int offset = cell_meta_t::offset_for_level(max_level());
    int nthreads = IRIS_CUDA_NTHREADS;
    int nblocks = IRIS_CUDA_NBLOCKS(nleafs, nthreads);
    k_distribute_particles<<<nblocks, nthreads>>>(in_particles, in_count, in_flags, out_target, offset, nleafs);

    // stack size is dubious, so let's see if it actually worked
    // cudaDeviceSynchronize();
    // IRIS_CUDA_CHECK_ERROR;
}


//////////////////
// Link parents //
//////////////////


__global__ void k_link_parents_proper(cell_t *io_cells, int start, int end)
{
    IRIS_CUDA_SETUP_WS(end-start);
    
    for(int j=start+from;j<start+to;j++) {
	if((io_cells[j].num_children != 0) ||                   // cell is a non-empty leaf
	   (io_cells[j].flags & IRIS_FMM_CELL_HAS_CHILDREN) ||  // or cell is a non-leaf and has some children
	   (io_cells[j].flags & IRIS_FMM_CELL_ALIEN_NL)) {        // or is an alien cell
	    int parent = cell_meta_t::parent_of(j);
	    atomicAdd(&io_cells[parent].flags, IRIS_FMM_CELL_HAS_CHILD1 << ((j - start) % 8));
	}
    }
}

__global__ void k_compute_ses_nl(cell_t *io_cells, int start, int end)
{
    IRIS_CUDA_SETUP_WS(end-start);
    for(int j=start+from;j<start+to;j++) {
	if(io_cells[j].ses.r != 0.0) {
	    continue;
	}
	sphere_t S[8];
	int ns = 0;
	for(int k = 0;k<8;k++) {
	    int mask = IRIS_FMM_CELL_HAS_CHILD1 << k;
	    if(io_cells[j].flags & mask) {
		int childID = end + 8*(j-start) + k;
		S[ns].c.r[0] = io_cells[childID].ses.c.r[0];
		S[ns].c.r[1] = io_cells[childID].ses.c.r[1];
		S[ns].c.r[2] = io_cells[childID].ses.c.r[2];
		S[ns].r = io_cells[childID].ses.r;
		ns++;
	    }
	}
	ses_of_spheres(S, ns, &(io_cells[j].ses));
    }
}
    
void fmm::link_parents_gpu(cell_t *io_cells)
{
    cudaDeviceSynchronize();

    //stack size is dubious, so let's see if it distribute_particles actually worked
    IRIS_CUDA_CHECK_ERROR;
    
    for(int i=max_level();i>0;i--) {
	int start = cell_meta_t::offset_for_level(i);
	int end = cell_meta_t::offset_for_level(i+1);
	int n = end - start;
	int nthreads = IRIS_CUDA_NTHREADS;
	int nblocks = IRIS_CUDA_NBLOCKS(n, nthreads);
	k_link_parents_proper<<<nthreads, nblocks>>>(io_cells, start, end);
    }

    for(int i=max_level()-1;i>=0;i--) {
    	int start = cell_meta_t::offset_for_level(i);
    	int end = cell_meta_t::offset_for_level(i+1);
    	int n = end - start;
    	int nthreads = IRIS_CUDA_NTHREADS;
    	int nblocks = IRIS_CUDA_NBLOCKS(n, nthreads);
	k_compute_ses_nl<<<nthreads, nblocks>>>(io_cells, start, end);
    }
}

#endif
