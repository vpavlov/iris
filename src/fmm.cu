
#include <hip/hip_runtime.h>
// -*- c++ -*-
//==============================================================================
// IRIS - Long-range Interaction Solver Library
//
// Copyright (c) 2017-2021, the National Center for Supercomputing Applications
//
// Primary authors:
//     Valentin Pavlov <vpavlov@rila.bg>
//     Peicho Petkov <peicho@phys.uni-sofia.bg>
//     Stoyan Markov <markov@acad.bg>
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
// THE SOFTWARE.
//==============================================================================
#ifdef IRIS_CUDA
#include <assert.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include "cuda.h"
#include "comm_rec.h"
#include "fmm.h"
#include "real.h"
#include "ses.h"
#include "fmm_kernels.h"
#include "fmm_pair.h"

using namespace ORG_NCSA_IRIS;


////////////////////
// Load Particles //
////////////////////


__global__ void k_load_charges(iris_real *charges, int ncharges, int hwm,
			       iris_real xlo, iris_real ylo, iris_real zlo,
			       iris_real lsx, iris_real lsy, iris_real lsz,
			       int max_level, int offset, particle_t *m_particles, int rank)
{
    IRIS_CUDA_SETUP_WS(ncharges);
    for(int i=from;i<to;i++) {
	iris_real tx = (charges[i * 5 + 0] - xlo) / lsx;
	iris_real ty = (charges[i * 5 + 1] - ylo) / lsy;
	iris_real tz = (charges[i * 5 + 2] - zlo) / lsz;
	
	int cellID = cell_meta_t::leaf_coords_to_ID(tx, ty, tz, max_level);
	int chargeID = (int)charges[i*5 + 4];
	
	m_particles[i+hwm].rank = rank;
	m_particles[i+hwm].index = chargeID;
	m_particles[i+hwm].cellID = cellID;
	m_particles[i+hwm].xyzq[0] = charges[i*5+0];
	m_particles[i+hwm].xyzq[1] = charges[i*5+1];
	m_particles[i+hwm].xyzq[2] = charges[i*5+2];
	m_particles[i+hwm].xyzq[3] = charges[i*5+3];
	m_particles[i+hwm].tgt[0] = 0.0;
	m_particles[i+hwm].tgt[1] = 0.0;
	m_particles[i+hwm].tgt[2] = 0.0;
	m_particles[i+hwm].tgt[3] = 0.0;
    }
}

__global__ void k_extract_cellID(particle_t *m_particles, int n, int *cellID_keys)
{
    IRIS_CUDA_SETUP_WS(n);
    cellID_keys[from] = m_particles[from].cellID;
}

void fmm::load_particles_gpu()
{
    // Find the total amount of local charges. This is just a simple sum of ncharges from all incoming rank.
    m_nparticles = 0;
    for(int rank = 0; rank < m_iris->m_client_size; rank++ ) {
	m_nparticles += m_iris->m_ncharges[rank];
    }

    // Allocate GPU memory for m_particles (and m_xparticles, it will be at the end of m_particles)
    // Also, allocate same sized array for the atom types (=1 if halo atom, =0 if own atom)
    m_particles = (particle_t *)memory::wmalloc_gpu_cap((void *)m_particles, m_nparticles, sizeof(particle_t), &m_npart_cap);
    m_keys = (int *)memory::wmalloc_gpu_cap((void *)m_keys, m_nparticles, sizeof(int), &m_keys_cap);

    // Allocate GPU memory for the charges coming from all client ranks
    // This is done all at once so as no to interfere with mem transfer/kernel overlapping in the next loop
    for(int rank = 0; rank < m_iris->m_client_size; rank++) {
	int ncharges = m_iris->m_ncharges[rank];
	iris_real *charges = m_iris->m_charges[rank];
	m_charges_gpu[rank] = (iris_real *)memory::wmalloc_gpu_cap((void *)m_charges_gpu[rank], ncharges, 5*sizeof(iris_real), &m_charges_gpu_cap[rank]);
    }

    // Start the particle loading itself. The <H2D, kernel> pairs runs on separate streams and thus overlaps memory transfer for more than 1 source rank
    int offset = cell_meta_t::offset_for_level(max_level());
    int nd = 1 << max_level();
    int hwm = 0;
    for(int rank = 0; rank < m_iris->m_client_size; rank++ ) {
	int ncharges = m_iris->m_ncharges[rank];
	iris_real *charges = m_iris->m_charges[rank];
	cudaMemcpyAsync(m_charges_gpu[rank], charges, ncharges * 5 * sizeof(iris_real), cudaMemcpyDefault, m_streams[rank % IRIS_CUDA_FMM_NUM_STREAMS]);
	
	int nthreads = MIN(IRIS_CUDA_NTHREADS, ncharges);
	int nblocks = IRIS_CUDA_NBLOCKS(ncharges, nthreads);
	k_load_charges<<<nblocks, nthreads, 0, m_streams[rank % IRIS_CUDA_FMM_NUM_STREAMS]>>>(m_charges_gpu[rank], ncharges, hwm,
											      m_domain->m_global_box.xlo, m_domain->m_global_box.ylo, m_domain->m_global_box.zlo,
											      m_leaf_size[0], m_leaf_size[1], m_leaf_size[2],
											      max_level(), offset, m_particles, rank);
	hwm += ncharges;
    }
        
    cudaDeviceSynchronize();  // all k_load_charges kernels must have finished to have valid m_particles

    // Get the cellIDs in the reordered array
    int nthreads = MIN(IRIS_CUDA_NTHREADS, m_nparticles);
    int nblocks = IRIS_CUDA_NBLOCKS(m_nparticles, nthreads);
    k_extract_cellID<<<nblocks, nthreads, 0, m_streams[0]>>>(m_particles, m_nparticles, m_keys);
    cudaStreamSynchronize(m_streams[0]);  // we want the above to finish before we do the sorts below
    
    // We now need to sort the m_particles by cellID
    thrust::device_ptr<int>         keys(m_keys);
    thrust::device_ptr<particle_t>  part(m_particles);
    thrust::sort_by_key(thrust::cuda::par.on(m_streams[0]), keys, keys+m_nparticles, part);
    
    m_logger->info("This rank owns %d particles", m_nparticles);
}


/////////////////////////////
// Distribute particles v1 //
/////////////////////////////


__device__ void d_compute_com(particle_t *in_particles, int num_points, int first_child, cell_t *out_target)
{
    // iris_real M = 0.0;
    // for(int i=0;i<num_points;i++) {
    // 	M += in_particles[first_child+i].xyzq[3];
    // }
    for(int i=0;i<num_points;i++) {
    	out_target->ses.c.r[0] += in_particles[first_child+i].xyzq[0]; // * in_particles[first_child+i].xyzq[3];
    	out_target->ses.c.r[1] += in_particles[first_child+i].xyzq[1]; // * in_particles[first_child+i].xyzq[3];
    	out_target->ses.c.r[2] += in_particles[first_child+i].xyzq[2]; // * in_particles[first_child+i].xyzq[3];
    }

    out_target->ses.c.r[0] /= num_points;
    out_target->ses.c.r[1] /= num_points;
    out_target->ses.c.r[2] /= num_points;

    iris_real max_dist2 = 0.0;
    for(int i=0;i<num_points;i++) {
	iris_real dx = in_particles[first_child+i].xyzq[0] - out_target->ses.c.r[0];
	iris_real dy = in_particles[first_child+i].xyzq[1] - out_target->ses.c.r[1];
	iris_real dz = in_particles[first_child+i].xyzq[2] - out_target->ses.c.r[2];
	iris_real dist2 = dx*dx + dy*dy + dz*dz;
	if(dist2 > max_dist2) {
	    max_dist2 = dist2;
	}
    }
    out_target->ses.r = sqrt(max_dist2);
}

__device__ int d_bsearch(particle_t *in_particles, int in_count, int cellID)
{
    int start = 0;
    int end = in_count-1;
    while (start <= end) {
        int m = start + (end - start) / 2;
        if (in_particles[m].cellID == cellID) {
            return m;
	}else if (in_particles[m].cellID < cellID) {
            start = m + 1;
	}else {
	    end = m - 1;
	}
    }
    return -1; 
}

__global__ void k_distribute_particles(particle_t *in_particles, int in_count, int in_flags, cell_t *out_target, int offset, int *m_max_particles_gpu)
{
    int tid = IRIS_CUDA_TID;
    int cellID = offset + tid;

    int from = d_bsearch(in_particles, in_count, cellID);
    if(from == -1) {
    	return;
    }
    int to = from;
    
    while(from > 0 && in_particles[from].cellID >= cellID)       { from--; }
    while(from < in_count && in_particles[from].cellID < cellID) { from++; }
    while(to < in_count-1 && in_particles[to].cellID <= cellID)  { to++; }
    while(to >= 0 && in_particles[to].cellID > cellID)           { to--; }
    
    int num_children = (to - from + 1);
    if(num_children <= 0) {
	return;
    }
    out_target[cellID].first_child = from;
    out_target[cellID].num_children = num_children;
    out_target[cellID].flags = in_flags;
    d_compute_com(in_particles, num_children, from, out_target+cellID);
    atomicMax(m_max_particles_gpu, num_children);
}


void fmm::distribute_particles_gpu_v1(particle_t *in_particles, int in_count, int in_flags, struct cell_t *out_target)
{
    if(in_count == 0) {
	return;
    }
    
    int nleafs = (1 << 3 * max_level());
    int offset = cell_meta_t::offset_for_level(max_level());
    int nthreads = MIN(IRIS_CUDA_NTHREADS, nleafs);
    int nblocks = IRIS_CUDA_NBLOCKS(nleafs, nthreads);
    k_distribute_particles<<<nblocks, nthreads, 0, m_streams[0]>>>(in_particles, in_count, in_flags, out_target, offset, m_max_particles_gpu);
    cudaMemcpyAsync(&m_max_particles, m_max_particles_gpu, sizeof(int), cudaMemcpyDefault, m_streams[0]); // TODO: make this async
}


//////////////////////////
// Distribute particles //
//////////////////////////


__global__ void k_init_first_child(cell_t *out_target, int offset)
{
    out_target[IRIS_CUDA_TID + offset].first_child = INT_MAX;
}


__global__ void k_find_range(particle_t *in_particles, int in_count, cell_t *out_target, int tile_size, int tile_offset, int in_flags)
{
    int i = IRIS_CUDA_TID;
    if(i >= tile_size) {
	return;
    }
    
    i += tile_offset;  // particle index
    if(i >= in_count) {
	return;
    }

    int cellID = in_particles[i].cellID;
    atomicMin(&(out_target[cellID].first_child), i);
    atomicAdd(&(out_target[cellID].num_children), 1);
    out_target[cellID].flags = in_flags;
 
    // prepare to find the center of mass
    atomicAdd(&(out_target[cellID].ses.c.r[0]), in_particles[i].xyzq[0]);
    atomicAdd(&(out_target[cellID].ses.c.r[1]), in_particles[i].xyzq[1]);
    atomicAdd(&(out_target[cellID].ses.c.r[2]), in_particles[i].xyzq[2]);
}

__global__ void k_find_max_particles(cell_t *out_target, int offset, int *m_max_particles_gpu)
{
    int tid = IRIS_CUDA_TID;
    int cellID = offset + tid;
    cell_t *leaf = &(out_target[cellID]);
    atomicMax(m_max_particles_gpu, leaf->num_children);
}


__global__ void k_find_ses(particle_t *in_particles, int in_flags, cell_t *out_target, int offset)
{
    int tid = IRIS_CUDA_TID;
    int cellID = offset + tid;
    cell_t *leaf = &(out_target[cellID]);
    int num_points = leaf->num_children;

    leaf->ses.c.r[0] /= num_points;
    leaf->ses.c.r[1] /= num_points;
    leaf->ses.c.r[2] /= num_points;

    iris_real max_dist2 = 0.0;
    for(int i=0;i<num_points;i++) {
    	iris_real dx = in_particles[leaf->first_child+i].xyzq[0] - leaf->ses.c.r[0];
    	iris_real dy = in_particles[leaf->first_child+i].xyzq[1] - leaf->ses.c.r[1];
    	iris_real dz = in_particles[leaf->first_child+i].xyzq[2] - leaf->ses.c.r[2];
    	iris_real dist2 = dx*dx + dy*dy + dz*dz;
    	if(dist2 > max_dist2) {
    	    max_dist2 = dist2;
    	}
    }
    leaf->ses.r = sqrt(max_dist2);
}

void fmm::distribute_particles_gpu(particle_t *in_particles, int in_count, int in_flags, struct cell_t *out_target)
{
    if(in_count == 0) {
	return;
    }

    // First, set first_child of all leafs to INT_MAX to facilitate MIN in the next kernel
    int nleafs = (1 << 3 * max_level());
    int offset = cell_meta_t::offset_for_level(max_level());
    int nthreads = MIN(IRIS_CUDA_NTHREADS, nleafs);
    int nblocks = IRIS_CUDA_NBLOCKS(nleafs, nthreads);
    k_init_first_child<<<nblocks, nthreads>>>(out_target, offset);

    // Then, find the first_child and num_children for each leaf
    // Also, sum all particle coordinates for each cell to prepare to find the center of mass
    // Do this in several streams to reduce atomic conflicts inside threads
    int nstreams = 4;
    int tile_offset;
    int tile_size = in_count / nstreams + ((in_count % nstreams)?1:0);
    dim3 nthreads2(IRIS_CUDA_NTHREADS, 1, 1);
    dim3 nblocks2((tile_size-1)/IRIS_CUDA_NTHREADS + 1, 1, 1);

    for(int i=0;i<nstreams;i++) {
	tile_offset = i * tile_size;
	k_find_range<<<nblocks2, nthreads2, 0, m_streams[i]>>>(in_particles, in_count, out_target, tile_size, tile_offset, in_flags);
    }
    cudaDeviceSynchronize();
    
    k_find_ses<<<nblocks, nthreads, 0, m_streams[0]>>>(in_particles, in_flags, out_target, offset);
    k_find_max_particles<<<nblocks, nthreads, 0, m_streams[1]>>>(out_target, offset, m_max_particles_gpu);
    cudaMemcpy(&m_max_particles, m_max_particles_gpu, sizeof(int), cudaMemcpyDefault);
}


//////////////////////////////
// Distribute xparticles v1 //
//////////////////////////////


__device__ int d_xbsearch(xparticle_t *in_particles, int in_count, int cellID)
{
    int start = 0;
    int end = in_count-1;
    while (start <= end) {
        int m = start + (end - start) / 2;
        if (in_particles[m].cellID == cellID) {
            return m;
	}else if (in_particles[m].cellID < cellID) {
            start = m + 1;
	}else {
	    end = m - 1;
	}
    }
    return -1; 
}

__global__ void k_distribute_xparticles(xparticle_t *in_particles, int in_count, int in_flags, cell_t *out_target, int offset, int *m_max_particles_gpu)
{
    int tid = IRIS_CUDA_TID;
    int cellID = offset + tid;
    int from = d_xbsearch(in_particles, in_count, cellID);
    if(from == -1) {
    	return;
    }
    int to = from;
    
    while(from > 0 && in_particles[from].cellID >= cellID)       { from--; }
    while(from < in_count && in_particles[from].cellID < cellID) { from++; }
    while(to < in_count-1 && in_particles[to].cellID <= cellID)  { to++; }
    while(to >= 0 && in_particles[to].cellID > cellID)           { to--; }
    
    int num_children = (to - from + 1);
    if(num_children <= 0) {
	return;
    }
    out_target[cellID].first_child = from;
    out_target[cellID].num_children = num_children;
    out_target[cellID].flags = in_flags;
    atomicMax(m_max_particles_gpu, num_children);
}

void fmm::distribute_xparticles_gpu_v1(xparticle_t *in_particles, int in_count, int in_flags, struct cell_t *out_target)
{
    if(in_count == 0) {
	return;
    }
    
    int nleafs = (1 << 3 * max_level());
    int offset = cell_meta_t::offset_for_level(max_level());
    int nthreads = MIN(IRIS_CUDA_NTHREADS, nleafs);
    int nblocks = IRIS_CUDA_NBLOCKS(nleafs, nthreads);
    k_distribute_xparticles<<<nblocks, nthreads, 0, m_streams[0]>>>(in_particles, in_count, in_flags, out_target, offset, m_max_particles_gpu);
    cudaMemcpyAsync(&m_max_particles, m_max_particles_gpu, sizeof(int), cudaMemcpyDefault, m_streams[0]); // TODO: make this async
}


///////////////////////////
// Distribute xparticles //
///////////////////////////


__global__ void k_find_xrange(xparticle_t *in_particles, int in_count, cell_t *out_target, int tile_size, int tile_offset, int in_flags)
{
    int i = IRIS_CUDA_TID;
    if(i >= tile_size) {
	return;
    }
    
    i += tile_offset;  // particle index
    if(i >= in_count) {
	return;
    }

    int cellID = in_particles[i].cellID;
    atomicMin(&(out_target[cellID].first_child), i);
    atomicAdd(&(out_target[cellID].num_children), 1);
    out_target[cellID].flags = in_flags;
}


void fmm::distribute_xparticles_gpu(xparticle_t *in_particles, int in_count, int in_flags, struct cell_t *out_target)
{
    if(in_count == 0) {
	return;
    }

    cudaDeviceSynchronize();
    
    // Then, find the first_child and num_children for each leaf
    // Also, sum all particle coordinates for each cell to prepare to find the center of mass
    // Do this in several streams to reduce atomic conflicts inside threads
    int nstreams = 4;
    int tile_offset;
    int tile_size = in_count / nstreams + ((in_count % nstreams)?1:0);
    dim3 nthreads2(IRIS_CUDA_NTHREADS, 1, 1);
    dim3 nblocks2((tile_size-1)/IRIS_CUDA_NTHREADS + 1, 1, 1);

    for(int i=0;i<nstreams;i++) {
	tile_offset = i * tile_size;
	k_find_xrange<<<nblocks2, nthreads2, 0, m_streams[i]>>>(in_particles, in_count, out_target, tile_size, tile_offset, in_flags);
    }
    cudaDeviceSynchronize();

    int nleafs = (1 << 3 * max_level());
    int offset = cell_meta_t::offset_for_level(max_level());
    int nthreads = MIN(IRIS_CUDA_NTHREADS, nleafs);
    int nblocks = IRIS_CUDA_NBLOCKS(nleafs, nthreads);
    k_find_max_particles<<<nblocks, nthreads, 0, m_streams[1]>>>(out_target, offset, m_max_particles_gpu);
    cudaMemcpy(&m_max_particles, m_max_particles_gpu, sizeof(int), cudaMemcpyDefault);
}


//////////////////
// Link parents //
//////////////////


__global__ void k_link_parents_proper(cell_t *io_cells, int start, int end)
{
    IRIS_CUDA_SETUP_WS(end-start);
    
    for(int j=start+from;j<start+to;j++) {
	if((io_cells[j].num_children != 0) ||                   // cell is a non-empty leaf
	   (io_cells[j].flags & IRIS_FMM_CELL_HAS_CHILDREN) ||  // or cell is a non-leaf and has some children
	   (io_cells[j].flags & IRIS_FMM_CELL_ALIEN_NL)) {        // or is an alien cell
	    int parent = cell_meta_t::parent_of(j);
	    atomicOr(&io_cells[parent].flags, IRIS_FMM_CELL_HAS_CHILD1 << ((j - start) % 8));
	}
    }
}

__global__ void k_compute_ses_nl(cell_t *io_cells, int start, int end)
{
    IRIS_CUDA_SETUP_WS(end-start);
    for(int j=start+from;j<start+to;j++) {
	if(io_cells[j].ses.r != 0.0) {
	    continue;
	}
	sphere_t S[8];
	int ns = 0;
	for(int k = 0;k<8;k++) {
	    int mask = IRIS_FMM_CELL_HAS_CHILD1 << k;
	    if(io_cells[j].flags & mask) {
		int childID = end + 8*(j-start) + k;
		S[ns].c.r[0] = io_cells[childID].ses.c.r[0];
		S[ns].c.r[1] = io_cells[childID].ses.c.r[1];
		S[ns].c.r[2] = io_cells[childID].ses.c.r[2];
		S[ns].r = io_cells[childID].ses.r;
		ns++;
	    }
	}
	ses_of_spheres(S, ns, &(io_cells[j].ses));
    }
}
    
void fmm::link_parents_gpu(cell_t *io_cells)
{
    for(int i=max_level();i>0;i--) {
	int start = cell_meta_t::offset_for_level(i);
	int end = cell_meta_t::offset_for_level(i+1);
	int n = end - start;
	int nthreads = MIN(IRIS_CUDA_NTHREADS, n);
	int nblocks = IRIS_CUDA_NBLOCKS(n, nthreads);
	k_link_parents_proper<<<nblocks, nthreads, 0, m_streams[0]>>>(io_cells, start, end);
    }

    for(int i=max_level()-1;i>=0;i--) {
    	int start = cell_meta_t::offset_for_level(i);
    	int end = cell_meta_t::offset_for_level(i+1);
    	int n = end - start;
    	int nthreads = MIN(IRIS_CUDA_NTHREADS, n);
    	int nblocks = IRIS_CUDA_NBLOCKS(n, nthreads);
	k_compute_ses_nl<<<nblocks, nthreads, 0, m_streams[0]>>>(io_cells, start, end);
    }
}


//////////////
// Eval M2M //
//////////////


__global__ void k_eval_m2m(cell_t *in_cells, bool invalid_only, int offset, int children_offset, iris_real *m_M, int m_nterms, int m_order)
{
    iris_real scratch[(IRIS_FMM_MAX_ORDER+1) * (IRIS_FMM_MAX_ORDER+1)];
    
    int tid = IRIS_CUDA_TID;
    int tcellID = tid + offset;
    int j = blockIdx.y;
    
    if(invalid_only && (in_cells[tcellID].flags & IRIS_FMM_CELL_VALID_M)) {
	return;
    }

    if(!(in_cells[tcellID].flags & (IRIS_FMM_CELL_HAS_CHILD1 << j))) {
	return;
    }
    
    iris_real cx = in_cells[tcellID].ses.c.r[0];
    iris_real cy = in_cells[tcellID].ses.c.r[1];
    iris_real cz = in_cells[tcellID].ses.c.r[2];
    
    iris_real *M = m_M + tcellID * m_nterms;
    
    int scellID = children_offset + 8*tid + j;
    iris_real x = in_cells[scellID].ses.c.r[0] - cx;
    iris_real y = in_cells[scellID].ses.c.r[1] - cy;
    iris_real z = in_cells[scellID].ses.c.r[2] - cz;
    
    m2m(m_order, x, y, z, m_M + scellID * m_nterms, M, scratch);
    in_cells[tcellID].flags |= IRIS_FMM_CELL_VALID_M;
}

void fmm::eval_m2m_gpu(cell_t *in_cells, bool invalid_only)
{
    cudaStreamSynchronize(m_streams[0]);  // wait for link parents
    cudaStreamSynchronize(m_streams[1]);  // wait for p2m
    int from, to;
    if(invalid_only) {
	from = m_local_root_level-1;
	to = 0;
    }else {
	from = max_level()-1;
	to = m_local_root_level;
    }
    
    for(int level = from;level>=to;level--) {
	int start = cell_meta_t::offset_for_level(level);
	int end = cell_meta_t::offset_for_level(level+1);
	int n = end - start;
	dim3 nthreads(MIN(IRIS_CUDA_NTHREADS, n), 1, 1);
	dim3 nblocks((n-1)/IRIS_CUDA_NTHREADS+1, 8, 1);
	k_eval_m2m<<<nblocks, nthreads, 0, m_streams[1]>>>(in_cells, invalid_only, start, end, m_M, m_nterms, m_order);
    }
}


////////////////////
// Relink parents //
////////////////////


__global__ void k_clear_nl_children(cell_t *io_cells, int count)
{
    int tid = IRIS_CUDA_TID;
    if(tid < count) {
	io_cells[tid].flags &= ~IRIS_FMM_CELL_HAS_CHILDREN;
    }
}

__global__ void k_clear_nl_ses(cell_t *io_cells, int count)
{
    int tid = IRIS_CUDA_TID;
    if(tid < count) {
	io_cells[tid].ses.r = 0.0;
    }
}


void fmm::relink_parents_gpu(cell_t *io_cells)
{
    int end = cell_meta_t::offset_for_level(max_level());
    int nthreads = MIN(IRIS_CUDA_NTHREADS, end);
    int nblocks = IRIS_CUDA_NBLOCKS(end, nthreads);
    k_clear_nl_children<<<nblocks, nthreads>>>(io_cells, end);

    end = cell_meta_t::offset_for_level(m_local_root_level);
    nthreads = MIN(IRIS_CUDA_NTHREADS, end);
    nblocks = IRIS_CUDA_NBLOCKS(end, nthreads);
    k_clear_nl_ses<<<nblocks, nthreads>>>(io_cells, end);

    link_parents_gpu(io_cells);
}


//////////////
// Eval M2L //
//////////////


__global__ void k_eval_m2l(interact_item_t *list, int list_size, cell_t *m_cells, cell_t *m_xcells, 
			   iris_real gxsize, iris_real gysize, iris_real gzsize, int m_nterms, int m_order, iris_real *m_M, iris_real *m_L)
{
    iris_real scratch[(IRIS_FMM_MAX_ORDER+1) * (IRIS_FMM_MAX_ORDER+1)];
    
    int tid = IRIS_CUDA_TID;
    if(tid >= list_size) {
	return;
    }

    int srcID = list[tid].sourceID;
    int destID = list[tid].targetID;
    iris_real xoff = list[tid].ix * gxsize;
    iris_real yoff = list[tid].iy * gysize;
    iris_real zoff = list[tid].iz * gzsize;
    
    iris_real sx = m_xcells[srcID].ses.c.r[0] + xoff;
    iris_real sy = m_xcells[srcID].ses.c.r[1] + yoff;
    iris_real sz = m_xcells[srcID].ses.c.r[2] + zoff;

    iris_real tx = m_cells[destID].ses.c.r[0];
    iris_real ty = m_cells[destID].ses.c.r[1];
    iris_real tz = m_cells[destID].ses.c.r[2];

    iris_real x = tx - sx;
    iris_real y = ty - sy;
    iris_real z = tz - sz;

    bool do_other_side = (list[tid].ix == 0 && list[tid].iy == 0 && list[tid].iz == 0);

    do_other_side = false;
    
    memset(scratch, 0, m_nterms*sizeof(iris_real));
    m2l_v2(m_order, x, y, z, m_M + srcID * m_nterms, m_L + destID * m_nterms, scratch,
	   m_M + destID * m_nterms, m_L + srcID * m_nterms, do_other_side);

    atomicOr(&(m_cells[destID].flags), IRIS_FMM_CELL_VALID_L);
    if(do_other_side) {
	atomicOr(&(m_cells[srcID].flags), IRIS_FMM_CELL_VALID_L);
    }
}

void fmm::eval_m2l_gpu()
{
    int n = m_m2l_list.size();
    if(n == 0) {
	return;
    }

    m_m2l_list_gpu = (interact_item_t *)memory::wmalloc_gpu_cap(m_m2l_list_gpu, n, sizeof(interact_item_t), &m_m2l_list_cap);
    cudaMemcpyAsync(m_m2l_list_gpu, m_m2l_list.data(), n * sizeof(interact_item_t), cudaMemcpyDefault, m_streams[0]);
    cudaEventRecord(m_m2l_memcpy_done, m_streams[0]);
    
    int nthreads = MIN(IRIS_CUDA_NTHREADS, n);
    int nblocks = IRIS_CUDA_NBLOCKS(n, nthreads);
    k_eval_m2l<<<nblocks, nthreads, 0, m_streams[0]>>>(m_m2l_list_gpu, n, m_cells, m_xcells, 
    						       m_domain->m_global_box.xsize, m_domain->m_global_box.ysize, m_domain->m_global_box.zsize, m_nterms,
    						       m_order, m_M, m_L);
    cudaEventSynchronize(m_m2l_memcpy_done);
    m_m2l_list.clear();
}


//////////////
// Eval L2L //
//////////////


__global__ void k_eval_l2l(cell_t *m_cells, int offset, int children_offset, iris_real *m_L, int m_nterms, int m_order)
{
    iris_real scratch[(IRIS_FMM_MAX_ORDER+1) * (IRIS_FMM_MAX_ORDER+1)];
    
    int tid = IRIS_CUDA_TID;
    int scellID = tid + offset;
    int j = blockIdx.y;

    if(scellID >= children_offset) {
	return;
    }
    
    if(!(m_cells[scellID].flags & IRIS_FMM_CELL_VALID_L)) {
	return;
    }

    if(!(m_cells[scellID].flags & (IRIS_FMM_CELL_HAS_CHILD1 << j))) {
	return;
    }
    
    iris_real cx = m_cells[scellID].ses.c.r[0];
    iris_real cy = m_cells[scellID].ses.c.r[1];
    iris_real cz = m_cells[scellID].ses.c.r[2];

    iris_real *L = m_L + scellID * m_nterms;
    
    int tcellID = children_offset + 8*tid + j;
    iris_real x = cx - m_cells[tcellID].ses.c.r[0];
    iris_real y = cy - m_cells[tcellID].ses.c.r[1];
    iris_real z = cz - m_cells[tcellID].ses.c.r[2];
    
    l2l(m_order, x, y, z, L, m_L + tcellID * m_nterms, scratch);
    m_cells[tcellID].flags |= IRIS_FMM_CELL_VALID_L;
}

void fmm::eval_l2l_gpu()
{
    for(int level = 0; level < m_depth-1; level++) {
	int start = cell_meta_t::offset_for_level(level);
	int end = cell_meta_t::offset_for_level(level+1);
	int n = end - start;
	dim3 nthreads(MIN(IRIS_CUDA_NTHREADS, n), 1, 1);
	dim3 nblocks((n-1)/IRIS_CUDA_NTHREADS+1, 8, 1);
	k_eval_l2l<<<nblocks, nthreads, 0, m_streams[0]>>>(m_cells, start, end, m_L, m_nterms, m_order);
    }
}


//////////////
// Eval L2P //
//////////////

__global__ void k_eval_l2p(cell_t *m_cells, int offset, particle_t *m_particles, int m_order, iris_real *m_L, int m_nterms)
{
    iris_real scratch[(IRIS_FMM_MAX_ORDER+1) * (IRIS_FMM_MAX_ORDER+1)];
    iris_real scratch_size = m_nterms * sizeof(iris_real);
    
    int leaf_idx = blockIdx.y * gridDim.z + blockIdx.z;   // Which cell we are processing
    int cellID = leaf_idx + offset;
    int j = IRIS_CUDA_TID;                                // Target particle inside cellID

    cell_t *leaf = m_cells + cellID;
    int npart = leaf->num_children;                       // Number of particles in the cell
     
    if(j >= npart || !(leaf->flags & IRIS_FMM_CELL_VALID_L)) {
	return;
    }

    particle_t *part = m_particles + leaf->first_child + j;

    iris_real *L = m_L + cellID * m_nterms;
    
    iris_real x = leaf->ses.c.r[0] - part->xyzq[0];
    iris_real y = leaf->ses.c.r[1] - part->xyzq[1];
    iris_real z = leaf->ses.c.r[2] - part->xyzq[2];
    iris_real q = part->xyzq[3];
    
    iris_real phi, Ex, Ey, Ez;
    
    l2p(m_order, x, y, z, q, L, scratch, &phi, &Ex, &Ey, &Ez);

    part->tgt[0] += phi;
    part->tgt[1] += Ex;
    part->tgt[2] += Ey;
    part->tgt[3] += Ez;
}

void fmm::eval_l2p_gpu()
{
    int offset = cell_meta_t::offset_for_level(max_level());
    int nleafs = m_tree_size - offset;
    
    dim3 nthreads(IRIS_CUDA_NTHREADS, 1, 1);
    dim3 nblocks((m_max_particles-1)/IRIS_CUDA_NTHREADS + 1, nleafs, 1);
    k_eval_l2p<<<nblocks, nthreads, 0, m_streams[0]>>>(m_cells, offset, m_particles, m_order, m_L, m_nterms);
}


///////////////////////////////
// Compute energy and virial //
///////////////////////////////


// TODO: compute virial
__global__ void k_compute_energy_and_virial(particle_t *m_particles, iris_real *out_ener, int npart)
{
    __shared__ iris_real ener_acc[IRIS_CUDA_NTHREADS];
    __shared__ iris_real vir_acc[IRIS_CUDA_NTHREADS][6];
    int iacc = threadIdx.x;
    ener_acc[iacc] = 0.0;
    
    int tid = IRIS_CUDA_TID;
    if(tid == 0) {
	*out_ener = 0.0;
    }
    if(tid < npart) {
    ener_acc[iacc] += m_particles[tid].tgt[0] * m_particles[tid].xyzq[3];
    iris_real xfx = m_particles[tid].xyzq[0] * m_particles[tid].tgt[1];
    iris_real yfx = m_particles[tid].xyzq[1] * m_particles[tid].tgt[1];
    iris_real zfx = m_particles[tid].xyzq[2] * m_particles[tid].tgt[1];
    iris_real xfy = m_particles[tid].xyzq[0] * m_particles[tid].tgt[2];
    iris_real yfy = m_particles[tid].xyzq[1] * m_particles[tid].tgt[2];
    iris_real zfy = m_particles[tid].xyzq[2] * m_particles[tid].tgt[2];
    iris_real xfz = m_particles[tid].xyzq[0] * m_particles[tid].tgt[3];
    iris_real yfz = m_particles[tid].xyzq[1] * m_particles[tid].tgt[3];
    iris_real zfz = m_particles[tid].xyzq[2] * m_particles[tid].tgt[3];
    vir_acc[iacc][0] += xfx;
    vir_acc[iacc][1] += yfy;
    vir_acc[iacc][2] += zfz;
    vir_acc[iacc][3] += (xfy + yfx);
    vir_acc[iacc][4] += (xfz + zfx);
    vir_acc[iacc][5] += (yfz + zfy);
    }

    __syncthreads();

    for(int i=blockDim.x; i>0; i/=2) {
	int stride = blockDim.x/i;
	if(iacc < (blockDim.x - stride) && iacc % (2*stride) == 0) {
        ener_acc[iacc] += ener_acc[iacc+stride];
        vir_acc[iacc][0] += vir_acc[iacc+stride][0];
        vir_acc[iacc][1] += vir_acc[iacc+stride][1];
        vir_acc[iacc][2] += vir_acc[iacc+stride][2];
        vir_acc[iacc][3] += vir_acc[iacc+stride][3];
        vir_acc[iacc][4] += vir_acc[iacc+stride][4];
        vir_acc[iacc][5] += vir_acc[iacc+stride][5];
	}
	__syncthreads();
    }
    if(iacc == 0) {
    atomicAdd(out_ener, ener_acc[0]);
    atomicAdd(out_ener+1, vir_acc[0][0]);
    atomicAdd(out_ener+2, vir_acc[0][1]);
    atomicAdd(out_ener+3, vir_acc[0][2]);
    atomicAdd(out_ener+4, vir_acc[0][3]);
    atomicAdd(out_ener+5, vir_acc[0][4]);
    atomicAdd(out_ener+6, vir_acc[0][5]);
    }
}

void fmm::compute_energy_and_virial_gpu()
{
    cudaDeviceSynchronize();
    
    int n = m_nparticles;
    int nthreads = MIN(IRIS_CUDA_NTHREADS, n);
    int nblocks = IRIS_CUDA_NBLOCKS(n, nthreads);
    k_compute_energy_and_virial<<<nblocks, nthreads, 0, m_streams[0]>>>(m_particles, m_evir_gpu, n);
    cudaMemcpyAsync(&(m_iris->m_Ek), m_evir_gpu, 7*sizeof(iris_real), cudaMemcpyDefault, m_streams[0]);
    cudaStreamSynchronize(m_streams[0]); // must be synchronous
    m_iris->m_Ek *= 0.5 * m_units->ecf;
    m_iris->m_virial[0] *= 0.5 * m_units->ecf;
	m_iris->m_virial[1] *= 0.5 * m_units->ecf;
	m_iris->m_virial[2] *= 0.5 * m_units->ecf;
	m_iris->m_virial[3] *= 0.25* m_units->ecf; //make the virial symmetric - multipling by extra 0.5 commumig from the averaging offdiagonal elementes 
	m_iris->m_virial[4] *= 0.25* m_units->ecf;
	m_iris->m_virial[5] *= 0.25* m_units->ecf;
}


//////////////////////
// Send back forces //
//////////////////////


__global__ void k_extract_rank(particle_t *m_particles, int n, int *keys)
{
    IRIS_CUDA_SETUP_WS(n);
    keys[from] = m_particles[from].rank;
}

void fmm::send_back_forces_gpu()
{
    thrust::device_ptr<int>         keys(m_keys);
    thrust::device_ptr<particle_t>  part(m_particles);

    int nthreads = MIN(IRIS_CUDA_NTHREADS, m_nparticles);
    int nblocks = IRIS_CUDA_NBLOCKS(m_nparticles, nthreads);
    k_extract_rank<<<nblocks, nthreads, 0, m_streams[0]>>>(m_particles, m_nparticles, m_keys);
    thrust::sort_by_key(thrust::cuda::par.on(m_streams[0]), keys, keys+m_nparticles, part);

    m_particles_cpu = (particle_t *)memory::wmalloc_gpu_cap(m_particles_cpu, m_nparticles, sizeof(particle_t), &m_particles_cpu_cap, true);
    cudaMemcpyAsync(m_particles_cpu, m_particles, m_nparticles * sizeof(particle_t), cudaMemcpyDefault, m_streams[0]);
    cudaStreamSynchronize(m_streams[0]); // must be sync
    send_back_forces_cpu(m_particles_cpu, false);
}

void fmm::cuda_specific_construct()
{
    for(int i=0;i<IRIS_CUDA_FMM_NUM_STREAMS;i++) {
	cudaStreamCreate(&m_streams[i]);
    }
    cudaEventCreate(&m_m2l_memcpy_done);
    cudaEventCreate(&m_p2p_memcpy_done);
    //cudaDeviceSetLimit(cudaLimitStackSize, 8192);  // otherwise distribute_particles won't work because of the welzl recursion
    cudaMalloc((void **)&m_evir_gpu, 7*sizeof(iris_real));
    cudaMalloc((void **)&m_max_particles_gpu, sizeof(int));

    // TODO: these must be deleted in a destruct (but not in fmm.cpp, which doesn't know about thrust potentially)
    m_a2a_cell_cnt_gpu = new thrust::device_vector<int>();
    m_a2a_cell_disp_gpu = new thrust::device_vector<int>();
    m_a2a_sendbuf_gpu = new thrust::device_vector<xparticle_t>();

    IRIS_CUDA_CHECK_ERROR;
}

#endif
