#include "fmm.h"
#include "real.h"

int main()
{
    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start, 0);
    //------------------------
    
    
    //------------------------
    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Elapsed time = %f ms\n", time);

    hipEventDestroy(start);
    hipEventDestroy(stop);
}
